#include "hip/hip_runtime.h"
//----------------------	EFM Finder simplifized architecture 
//----------------------	Mona Arabzadeh
//----------------------	Dey 1395  
//----------------------	PhD -- part3

//auto complete ==> ctlr + space

//nvcc -deviceemu	

#include <stdio.h>
#include <math.h>
#include<sstream>
#include <iostream>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <string>
#include "assert.h"
#include <conio.h>
#include <time.h>
#include <vector>
#include <algorithm>
#include <bitset>

#include <iterator> 

#include "cuPrintf.cuh"
#include "cuPrintf.cu"

#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""

#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

/*#include <thrust/device_vector.h>
#include <thrust/copy.h>*/

#include "structs.h"
#include "funcs.cpp"
#include "lock.h"


using namespace std;  //introduces namespace std

//-------------------------------------------------------------------------------------------------------------------------
//-------------------------------------------------------------------------------------------------------------------------
// read network from matrix 
// --- statics
// network is a set of NODEs 
// NODEs are consist of INPUTs and OUTPUTs 
// NODE dynamics: seen, FWorBW

// --- dynamics 
// PATH : flux, history(meta,pathnum), direction,    
//-------------------------------------------------------
__device__ unsigned int RNG()
{   
    unsigned int m_w = 150;
    unsigned int m_z = 40;

	unsigned int res = 0;

   // for(int i=0; i < 100; i++)
    //{
        m_z = 36969 * (m_z & 65535) + (m_z >> 16);
        m_w = 18000 * (m_w & 65535) + (m_w >> 16);

 //       cout <<(m_z << 16) + m_w << endl;  /* 32-bit result */
    //}

	res = ((m_z << 16) + m_w);
	return res;
}
//-------------------------------------------------------
__device__ void convDecimalToBase(int decimalNum, int BN, int* BS, int BSSize)
{
	//int* b = new int[BN];
	//memset(b, 0, BN);

	int x = decimalNum;
	int y = BN; 
	int point = BSSize - 1;

	if (x<y)
	{
		BS[point] = x;
	}
	else{
		while(x >= y)
		{
			BS[point] = x%y;
			x /= y; 
			point--;
		}

		BS[point] = x;
	}

	return; 
}
//-------------------------------------------------------
__device__ int findReacOrder(int reactionName, INPUT_ARRAY* inp)
{
	int ord = -1;

	for (int i=0; i<MaxNumOfRecInOut; i++)
	{
		if(inp[i].reacNum == reactionName)
		{
			ord = i;
			break;
		}
	}

	return ord;
}
//-------------------------------------------------------
__global__ void	depthFUNC(NET *d_network)
{
	int iMETA	=	blockIdx.x;					// on metabolites
	int	jThread	=	threadIdx.x;	

	int  baseStr[MaxDepth] = {0}; 
	int  baseNum = MaxNumOfRecInOut;

	convDecimalToBase(jThread, baseNum, baseStr, MaxDepth);

	int isDone		=	0;		//set if reach level = 10 || see iMETA 

	int firstRec	= baseStr[0];

	int currentNODE		[STACKSIZE];
	int currentLevel	[STACKSIZE];

	int stackSize		= 0;			//keep the size of elements which are needed to analyze 
	int stackPointer	= 0;			//just go forward when an element added

	currentNODE[0]		= iMETA;
	currentLevel[0]		= 1;
	stackSize++;

	int recSizeOut		=   (d_network[0].net[iMETA].METstatus & 0x000000F0) >> 4;
	int recSizeIn		=   (d_network[0].net[iMETA].METstatus & 0x0000000F);
	int recSizeInOut	=	0;

	if ((recSizeOut == 1) && (recSizeIn ==1)) //if it has only one input and only one output
		recSizeInOut = 1;

	while (stackSize>0)
	{
		int popCurrentNode  = currentNODE[stackPointer];
		int popCurrentLevel = currentLevel[stackPointer];

		if (stackPointer == 99)
			return;
		//cuPrintf("here");

		stackPointer++;
		stackSize--;

		int currentREC	= baseStr[popCurrentLevel-1]; // (jThread%MaxNumOfRecInOut);
		
		int ifLastMet	=   (d_network[0].net[popCurrentNode].outputs[currentREC].RECstatus & 0x0000FF00) >> 8;
		recSizeOut		=   (d_network[0].net[popCurrentNode].METstatus & 0x000000F0) >> 4;


		if(ifLastMet>0)
		{ //----------------------------------------------------------------------------------last metabolite :: the output reaction has no metabolite  

		//if(jThread==8)
		//	cuPrintf("CuuRec = %d;recSizeOut = %d\n\r", currentREC,recSizeOut);

			if (currentREC < recSizeOut) //if valid
			{
			//	cuPrintf("CuuRec = %d;recSizeOut = %d\n\r", currentREC,recSizeOut);
				int metSize = ifLastMet;//(d_network[0].net[popCurrentNode].outputs[currentREC].RECstatus & 0x0000FF00) >> 8; //??? check
				for (int i=0; i</*MaxNumOfMetInOut*/metSize; i++)
				{
					if (d_network[0].net[popCurrentNode].outputs[currentREC].metabolitNamesOut[i] == iMETA)
					{
						//set what is needed to be set
						isDone = 1;
						int ord = findReacOrder(d_network[0].net[popCurrentNode].outputs[currentREC].reacNum, d_network[0].net[iMETA].inputs);

						if (recSizeInOut == 0) {
							d_network[0].net[iMETA].inputs[ord].notGoodPrimaryCandida			= 1;
							d_network[0].net[iMETA].inputs[ord].reactionNameWeGotBackTo			= d_network[0].net[iMETA].outputs[firstRec].reacNum;
							//cuPrintf("META = %d;thread = %d; %d\n\r", iMETA,jThread,d_network[0].net[iMETA].outputs[firstRec].reacNum);

							d_network[0].net[iMETA].outputs[firstRec].notGoodPrimaryCandida		= 1;
							d_network[0].net[iMETA].outputs[firstRec].reactionNameWeGotBackTo	= d_network[0].net[iMETA].inputs[ord].reacNum;
							//cuPrintf("META = %d;thread = %d; %d\n\n\r", iMETA,jThread,d_network[0].net[iMETA].inputs[ord].reacNum);
						}
					}//endif we are done
					else if (popCurrentLevel <= MaxDepth) // full stack while  (level <= MaxDepth)
					{
						currentNODE	[stackPointer+stackSize] = d_network[0].net[popCurrentNode].outputs[currentREC].metabolitNamesOut[i];
						currentLevel[stackPointer+stackSize] = popCurrentLevel+1;
						stackSize++;
					}// end else push back nodeName nd level to stack

				}//endfor i:MaxNumOfMetInOut

				if(isDone == 1)
					break; //break while  if Done

			}// end if
			else
				break; // break the while -- not a valid reaction

		}//-----------------------------------------------------------------------------end if las metabolite (no reaction)
	}//end while


	//cuPrintf("Block = %d;thread = %d\n\r", iMETA,jThread);

}// end depthFUNC
//-------------------------------------------------------
//-------------------------------------------------------
__global__ void MetaINIT(NET *d_network, oneEFM *d_EFMs)
{
	int j = blockIdx.x;   // on candidates
	int k = threadIdx.x;  // on reactions

	//for (int j=0; j<NumberOfCandidates; j++)
	//{
		d_EFMs[j].recFlux[NumberOfREACTIONSsPlus-1] = 0;  // as an initial value in initialize ==> 0 :: notDone

		// set the first reaction flux and trig output and inputs 
		int firstReaction	= 0;									//sample :: 15			S2::0		S2::4
		//int firstMetabolite = 0;									//sample :: 15			S2::2		S2::2
		//for (int k=0; k<NumberOfREACTIONSs; k++)
		//{
			d_EFMs[j].recFlux[k] = 0;
		//}

		d_EFMs[j].recFlux[firstReaction] = 1;

		/*firstReaction	= 3;	
		d_EFMs[j].recFlux[firstReaction] = 1;

		firstReaction	= 8;	
		d_EFMs[j].recFlux[firstReaction] = 1;*/

	//}//endForThread
}
//---------------------------------------------------------
__device__ void wait()
{
	for (int i=0; i<10000; i++){}	
}
//----------------------------------------------------------
//----------------------------------------------------------
//----------------------------------------------------------
__global__ void METAx(NET *d_network, oneEFM *d_EFMs, int *parVal) // input :: the whole network, index ==> i :: Executation of METAx on i
{
	
	if (threadIdx.x == 0)
		__shared__ oneEFM test;// = d_EFMs[blockIdx.x];	//for sharedMEmory test ==> LATER

	while(1)
	{
		//...............................................................................
		int iMET=-1,jBLOCK=-1; 
		float extraFL = 0;
		//...............................................................................

		iMET	= threadIdx.x;					// on metabolites
		jBLOCK	= blockIdx.x;					// on blocks
		//...............................................................................
		int numberOfInputs		=	 d_network[0].net[iMET].METstatus & 0x0000000F;
		int numberOfOutputs		=   (d_network[0].net[iMET].METstatus & 0x000000F0) >> 4;
		//...............................................................................
		//if(iMET == 7)
		//	cuPrintf("exFlux = %f\n\r",extraFL);

		 //float ffll = 0;
		 //float ccff = 0; 
		int activeInpCnt = 0, activeOutCnt = 0;

calAgain:for (int fin=0; fin < numberOfInputs; fin++)
		 {
			float fl = d_EFMs[jBLOCK].recFlux[d_network[0].net[iMET].inputs[fin].reacNum];
			if (fl != 0)
				activeInpCnt++;
			 extraFL += fl/*d_EFMs[jBLOCK].recFlux[d_network[0].net[iMET].inputs[fin].reacNum]*/ * d_network[0].net[iMET].inputs[fin].Coef;
			 d_EFMs[jBLOCK].isUpdate[d_network[0].net[iMET].inputs[fin].reacNum] = 0;
		 }
		 for (int fin=0; fin < numberOfOutputs; fin++)
		 {
			 float fl = d_EFMs[jBLOCK].recFlux[d_network[0].net[iMET].outputs[fin].reacNum];
			 if (fl != 0)
				activeOutCnt++;
			 extraFL += fl/*d_EFMs[jBLOCK].recFlux[d_network[0].net[iMET].outputs[fin].reacNum]*/ * d_network[0].net[iMET].outputs[fin].Coef;
			 d_EFMs[jBLOCK].isUpdate[d_network[0].net[iMET].outputs[fin].reacNum] = 0;
		 }

		 //................................................................................
		 if (extraFL == 0)		
		 {
			 d_EFMs[jBLOCK].AllStable[iMET] = 0;
		 }//................................/endelse extraFL=0/............................
		 else if (extraFL > 0)
		 {
			 /*cuPrintf("extraFL > 0 \r\n");
			 parVal[0]++;
			 cuPrintf("step=%d", parVal[0]);*/

			 //cuPrintf("iMET = %d..... extraFL > 0", iMET);

			 d_EFMs[jBLOCK].AllStable[iMET] = 1;
			 //if nonVisited :: checkVisited go from Primaries
			 if(d_EFMs[jBLOCK].metVisited[iMET] == 0)
			 {
				 int tId = blockIdx.x + threadIdx.x;

				 int primaryCandidateInp = -1;
				 for (int fin=0; fin < numberOfInputs; fin++)
				 { if (d_EFMs[jBLOCK].recFlux[d_network[0].net[iMET].inputs[fin].reacNum] != 0) {primaryCandidateInp = fin; break;} }

				 //-------------------------------- go randomly out 
				 //unsigned int rnd = RNG();
				 
				 //commented for test
				 int outputITis		=tId %numberOfOutputs;//tId*(numberOfOutputs-1)%numberOfOutputs;//tId*(numberOfOutputs-1)%numberOfOutputs;
//				 int outputITis = 0; 
				 //if (iMET == 17)
				 //	 outputITis = 1;
				 int outputITisName	= d_network[0].net[iMET].outputs[outputITis].reacNum;

				  //for debug
				 /*if (iMET == 0)
					 outputITisName = 1;
				 else if(iMET == 1)
					 outputITisName = 3;
				 else if (iMET == 2)
					 outputITisName = 4;
				 else if (iMET == 4)
					 outputITisName = 7;
				 else if (iMET == 5)
					 outputITisName = 8;
				 else if (iMET == 6)
					 outputITisName = 9;
				 else if (iMET == 7)
					 outputITisName = 10;
				 else if (iMET == 8)
					 outputITisName = 11;
				 else if (iMET == 10)
					 outputITisName = 8;

				 for (int q=0; q<numberOfOutputs; q++)
				 { if (outputITisName == (d_network[0].net[iMET].outputs[q].reacNum))	{outputITis = q; break; }  }*/
				 //for debug


				 float newFlux = abs(extraFL/d_network[0].net[iMET].outputs[outputITis].Coef); 
				 /*if (d_EFMs[jBLOCK].recFlux[outputITisName] != 0 ){
				 if (d_EFMs[jBLOCK].recFlux[outputITisName] !=  newFlux)
				 {d_EFMs[jBLOCK].recFlux[NumberOfREACTIONSsPlus-1] = 2;}	//-- check as nonEFM	:: fluxes dont match	
				 }
				 else*/ 
				 if (d_EFMs[jBLOCK].isUpdate[outputITisName] == 0){
					 d_EFMs[jBLOCK].recFlux[outputITisName] = newFlux;
					 //cuPrintf("recName = %d;recFlux = %f\n\r", outputITisName,newFlux);
					 d_EFMs[jBLOCK].isUpdate[outputITisName] = 1;
				 }
				 else
				 {
					  //cuPrintf("washere");
					 goto calAgain;
				 }
				 //wait();
				 //-------------------------------- set primary input and output (primaryInput is a challenge => check one of them)
				 d_EFMs[jBLOCK].primaryInput[iMET]  = primaryCandidateInp;
				 d_EFMs[jBLOCK].primaryOutput[iMET] = outputITis;
				 //--------------------------------
				 d_EFMs[jBLOCK].metVisited[iMET] = 1;
			 }//.........................................................end if nonVisited

			 else if (d_EFMs[jBLOCK].metVisited[iMET] == 1)                           //   ==0 or the same as primary reaction  :: nonEFM
			 {
				 if(((d_network[0].net[iMET].outputs[d_EFMs[jBLOCK].primaryOutput[iMET]].notGoodPrimaryCandida == 0) && (d_EFMs[jBLOCK].loop[iMET] < LOOPLimit)) || (activeInpCnt == 1)/*d_EFMs[jBLOCK].loop[iMET] == 0*/) //from primaries - out
				 {
					 int primryOutName = d_network[0].net[iMET].outputs[d_EFMs[jBLOCK].primaryOutput[iMET]].reacNum;
					 float oldFlux = d_EFMs[jBLOCK].recFlux[primryOutName];
					 float c		  = abs(d_network[0].net[iMET].outputs[d_EFMs[jBLOCK].primaryOutput[iMET]].Coef);
					 if (d_EFMs[jBLOCK].isUpdate[primryOutName]==0){
						 d_EFMs[jBLOCK].recFlux[primryOutName] = ((oldFlux*c) + extraFL)/c;
						 //cuPrintf("recName = %d;recFlux = %f\n\r", primryOutName, d_EFMs[jBLOCK].recFlux[primryOutName]);
						 d_EFMs[jBLOCK].isUpdate[primryOutName] = 1;
					 }
					 else 
					 {
						 //cuPrintf("washere");
						 goto calAgain;
					 }
					 //wait();
					 d_EFMs[jBLOCK].loop[iMET]++; 
				 }
				 else if((d_network[0].net[iMET].outputs[d_EFMs[jBLOCK].primaryOutput[iMET]].notGoodPrimaryCandida == 1) && (d_EFMs[jBLOCK].loop[iMET] < LOOPLimit)/*(d_EFMs[jBLOCK].loop[iMET] > 0) && (d_EFMs[jBLOCK].loop[iMET] <= 2)*/)	//change direction
				 {
					 int primryInpName = d_network[0].net[iMET].inputs[d_EFMs[jBLOCK].primaryInput[iMET]].reacNum;
					 float oldFlux = d_EFMs[jBLOCK].recFlux[primryInpName];
					 float c		  = abs(d_network[0].net[iMET].inputs[d_EFMs[jBLOCK].primaryInput[iMET]].Coef);

					 float newFlux = abs(((oldFlux*c) - extraFL)/c);
					 /*if ((newFlux == oldFlux) || (newFlux == 0))
					 d_EFMs[jBLOCK].recFlux[NumberOfREACTIONSsPlus-1] = 2;
					 else*/
					 if(d_EFMs[jBLOCK].isUpdate[primryInpName] ==0){
						 d_EFMs[jBLOCK].recFlux[primryInpName] = newFlux;
						  //cuPrintf("recName = %d;recFlux = %f\n\r", primryInpName, newFlux);
						 d_EFMs[jBLOCK].isUpdate[primryInpName] = 1;
					 }
					 else
					 {
						 //cuPrintf("washere");
						 goto calAgain;
					 }
					 //wait();	
					 d_EFMs[jBLOCK].loop[iMET]++;
				 }
				 else if(d_EFMs[jBLOCK].loop[iMET] >= LOOPLimit)	//direction has changed before -- check as nonEFM (loop detected)
				 {
					 d_EFMs[jBLOCK].recFlux[NumberOfREACTIONSsPlus-1] = 2;
				 }

			 }//.........................................................end else visited
			 /*for (int test=0; test<NumberOfREACTIONSs; test++)
			 {
				 cuPrintf("%f\t",d_EFMs[jBLOCK].recFlux[test]);
			 }
			 cuPrintf("\r\n");
			 */
		 }//................................/endelse extraFL>0/............................
		 else if (extraFL < 0)
		 {
			 /*cuPrintf("extraFL < 0\r\n");
			 parVal[0]++;
			 cuPrintf("step=%d", parVal[0]);*/

			 extraFL = -extraFL;
			 d_EFMs[jBLOCK].AllStable[iMET] = 1;
			 //if nonVisited :: checkVisited go from Primaries
			 if(d_EFMs[jBLOCK].metVisited[iMET] == 0)
			 {
				 int tId = blockIdx.x + threadIdx.x;

				 int primaryCandidateOut = -1;
				 for (int fin=0; fin < numberOfOutputs; fin++)
				 { if (d_EFMs[jBLOCK].recFlux[d_network[0].net[iMET].outputs[fin].reacNum] != 0) {primaryCandidateOut = fin; break;} }

				 //-------------------------------- go randomly in
				 //unsigned int rnd = RNG();
				 int inputITis		= tId%numberOfInputs;//tId*(numberOfInputs-1)%numberOfInputs;//tId*(numberOfInputs-1)%numberOfInputs;
				 int inputITisName	= d_network[0].net[iMET].outputs[inputITis].reacNum;


				 //for debug
				/* if (iMET == 0)
					 inputITisName = 0;
				 else if(iMET == 1)
					 inputITisName = 1;
				 else if (iMET == 2)
					 inputITisName = 3; //1or3
				 else if (iMET == 4)
					 inputITisName = 4;
				 else if (iMET == 5)
					 inputITisName = 7;
				 else if (iMET == 6)
					 inputITisName = 8;
				 else if (iMET == 7)
					 inputITisName = 9;
				 else if (iMET == 8)
					 inputITisName = 10;
				 else if (iMET == 10)
					 inputITisName = 11;

				 for (int q=0; q<numberOfInputs; q++)
				 { if (inputITisName == (d_network[0].net[iMET].inputs[q].reacNum))	{inputITis = q; break; }  }*/

				 //for debug


				 float newFlux = abs(extraFL/d_network[0].net[iMET].outputs[inputITis].Coef); 
				 /*if (d_EFMs[jBLOCK].recFlux[inputITisName] != 0 ){
				 if (d_EFMs[jBLOCK].recFlux[inputITisName] !=  newFlux)
				 {d_EFMs[jBLOCK].recFlux[NumberOfREACTIONSsPlus-1] = 2;}	//-- check as nonEFM	:: fluxes dont match	
				 }
				 else */
				 if (d_EFMs[jBLOCK].isUpdate[inputITisName]==0){
					 d_EFMs[jBLOCK].recFlux[inputITisName] = newFlux;
					 //cuPrintf("recName = %d;recFlux = %f\n\r", inputITisName, newFlux);
					 d_EFMs[jBLOCK].isUpdate[inputITisName] = 1;
				 }
				 else 
				 {
					  //cuPrintf("washere");
					 goto calAgain;
				 }
				 //wait();
				 //-------------------------------- set primary input and output (primaryOutput is a challenge => check one of them)
				 d_EFMs[jBLOCK].primaryInput[iMET]  = inputITis;
				 d_EFMs[jBLOCK].primaryOutput[iMET] = primaryCandidateOut;
				 //--------------------------------
				 d_EFMs[jBLOCK].metVisited[iMET] = 1;
			 }//.........................................................end if nonVisited

			 else if (d_EFMs[jBLOCK].metVisited[iMET] == 1)                           //   ==0 or the same as primary reaction  :: nonEFM
			 {
				 if(((d_network[0].net[iMET].inputs[d_EFMs[jBLOCK].primaryInput[iMET]].notGoodPrimaryCandida==0) && (d_EFMs[jBLOCK].loop[iMET] < LOOPLimit)) || (activeOutCnt == 1)/*d_EFMs[jBLOCK].loop[iMET] == 0*/)		//from primaries - in
				 {
					 int primryInpName = d_network[0].net[iMET].inputs[d_EFMs[jBLOCK].primaryInput[iMET]].reacNum;

					 float oldFlux = d_EFMs[jBLOCK].recFlux[primryInpName];
					 float c		  = abs(d_network[0].net[iMET].inputs[d_EFMs[jBLOCK].primaryInput[iMET]].Coef);
					 if (d_EFMs[jBLOCK].isUpdate[primryInpName] == 0)
					 {
						 d_EFMs[jBLOCK].recFlux[primryInpName] = ((oldFlux*c) + extraFL)/c;
						 //cuPrintf("recName = %d;recFlux = %f\n\r", primryInpName,  d_EFMs[jBLOCK].recFlux[primryInpName]);
						 d_EFMs[jBLOCK].isUpdate[primryInpName] = 1 ;
					 }
					 else{
						//cuPrintf("washere");
						goto calAgain;
					 }
					 //wait();
					 d_EFMs[jBLOCK].loop[iMET]++; 
				 }
				 else if((d_network[0].net[iMET].inputs[d_EFMs[jBLOCK].primaryInput[iMET]].notGoodPrimaryCandida==1) && (d_EFMs[jBLOCK].loop[iMET] < LOOPLimit))/*(d_EFMs[jBLOCK].loop[iMET] > 0) && (d_EFMs[jBLOCK].loop[iMET] <= 2)*/	//change direction
				 {
					 int primryOutName = d_network[0].net[iMET].outputs[d_EFMs[jBLOCK].primaryOutput[iMET]].reacNum;

					 float oldFlux = d_EFMs[jBLOCK].recFlux[primryOutName];
					 float c		  = abs(d_network[0].net[iMET].outputs[d_EFMs[jBLOCK].primaryOutput[iMET]].Coef);
					 float newFlux = abs(((oldFlux*c) - extraFL)/c);
					 /*if ((newFlux == oldFlux) || (newFlux == 0))
					 d_EFMs[jBLOCK].recFlux[NumberOfREACTIONSsPlus-1] = 2;
					 else*/
					 if (d_EFMs[jBLOCK].isUpdate[primryOutName] == 0)
					 {
						 d_EFMs[jBLOCK].recFlux[primryOutName] = newFlux;
						 //cuPrintf("recName = %d;recFlux = %f\n\r", primryOutName, newFlux);
						 d_EFMs[jBLOCK].isUpdate[primryOutName] = 1;
					 }
					 else 
					 {
						// cuPrintf("washere");
						 goto calAgain;
					 }
					 //wait();

					 d_EFMs[jBLOCK].loop[iMET]++;
				 }
				 else if(d_EFMs[jBLOCK].loop[iMET] >= LOOPLimit)	//direction has changed before -- check as nonEFM (loop detected)
				 {
					 d_EFMs[jBLOCK].recFlux[NumberOfREACTIONSsPlus-1] = 2;
				 }

			 }//.........................................................end else visited
			 /*for (int test=0; test<NumberOfREACTIONSs; test++)
			 {
				 cuPrintf("%f\t",d_EFMs[jBLOCK].recFlux[test]);
			 }
			  cuPrintf("\r\n");*/
		 }//................................/endelse extraFL<0/............................
		 //................................................................................

		 //................................................................................
		 //...........................//All Threads Done Check//...........................
		 int AllEFMsDone = 1;

		 int check = 1;
		 if ((d_EFMs[jBLOCK].recFlux[NumberOfREACTIONSsPlus-1] == 0) && (d_EFMs[jBLOCK].recFlux[NumberOfREACTIONSsPlus-1] != 2))
		 {
			 for (int m=0; m<NumberOfMETABOLITEs; m++)
			 {
				 if (d_EFMs[jBLOCK].AllStable[m] == 1)
					 check = 0;
			 }
		 }
		 if (check == 1){
			 d_EFMs[jBLOCK].AllStableCount++;
			 //d_EFMs[jBLOCK].recFlux[NumberOfREACTIONSsPlus-1] = 1;
		 }
		 if(d_EFMs[jBLOCK].AllStableCount==8)
			 d_EFMs[jBLOCK].recFlux[NumberOfREACTIONSsPlus-1] = 1;

		 //--------------
		 for (int j=0; j<NumberOfCandidates; j++)
		 {
			 if (d_EFMs[j].recFlux[NumberOfREACTIONSsPlus-1] == 0)
				 AllEFMsDone = 0;
		 }
		 //---
		 if (AllEFMsDone) // all threads done
			 break;		// break while ==> copyback EFMs to main Memory

		 //__syncthreads();
		 //................................................................................
	}//end while 1

} //end METAx
//------------------------------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------
hipError_t EFM()
{
	hipError_t cudaStatus;

	oneEFM		*d_EFMs;		//[NumberOfThreads];
	NET			*d_network;

	int			*d_parVal;
	int			parVal[1]	= {0};

	// define device variables  

	hipDeviceReset();
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	//-----------------------------------------------------------------------------
	//=========================================================================== show memory usage of GPU
	size_t free_byte;
	size_t total_byte;
	cudaStatus = hipMemGetInfo(&free_byte, &total_byte);

	if ( hipSuccess != cudaStatus ){
		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cudaStatus) );
		exit(1);
	}

	double free_db = (double)free_byte; 
	double total_db = (double)total_byte;
	double used_db = total_db - free_db;

	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
		used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
	//==========================================================================
	//-----------------------------------------------------------------------------  copy memory to gpu
		//---
	cudaStatus = hipMalloc((void**)&d_parVal, sizeof(int));	
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}

	cudaStatus = hipMemcpy(d_parVal, &parVal, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//----


	int sizeE = sizeof EFMs;									//for test -- passed the test
	cudaStatus = hipMalloc((void**)&d_EFMs, sizeE);			//for test -- passed the test

	int sizeN = sizeof network;
	cudaStatus = hipMalloc((void**)&d_network, sizeN);	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	cudaStatus = hipMemcpy(d_network, &network, sizeN, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	/*cudaStatus = hipMemcpy(d_DONE, &DONE, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
	goto Error;
	}*/

	//=========================================================================== show memory usage of GPU
	//   size_t free_byte;
	//   size_t total_byte;
	cudaStatus = hipMemGetInfo(&free_byte, &total_byte);

	if ( hipSuccess != cudaStatus ){
		printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cudaStatus) );
		exit(1);
	}

	free_db = (double)free_byte; 
	total_db = (double)total_byte;
	used_db = total_db - free_db;

	printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
		used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
	//==========================================================================
	//-----------------------------------------------------------------------------
	//-----------------------------------------------------------------------------
	//-----------------------------------------------------------------------------
	//-----------------------------------------------------------------------------
	//-------------------------calculate related reactions  
	cudaPrintfInit ();

	//dim3 dimGrid0(NumberOfMETABOLITEs);		//each block == one metbolite
	//float x = MaxNumOfRecInOut; float y = MaxDepth; 
	//int maxPathNumber = pow(x, y);
	//dim3 dimBlock0(maxPathNumber);						//for each metbolite create all poibilities on threads ==> how many threads we nead -- how to calc indexes
	
	//depthFUNC<<<dimGrid0, dimBlock0>>> (d_network);

	//-------------------------
	//-------------------------


	dim3 dimGrid1(NumberOfCandidates);		
	dim3 dimBlock1(NumberOfREACTIONSs);

	MetaINIT<<<dimGrid1, dimBlock1>>>(d_network, d_EFMs);	

	hipProfilerStart(); 
	
	// number of blocks == number of metabolits :: chejuri ru block haye mokhtalef yek function ro active konam?
	dim3 dimGrid2(NumberOfCandidates);		//NumberOfMETABOLITEs = NumberOfBlocks
	dim3 dimBlock2(NumberOfMETABOLITEs);		//NumberOfThreads

	METAx<<<dimGrid2, dimBlock2>>>(d_network, d_EFMs, d_parVal/*d_EFMsss,*/ /*,d_hora*/);	//Numofblocks = NumberOfMETABOLITEs	BlockSize = numOfThreads = 10 (masalan)

	hipProfilerStop();

	cudaPrintfDisplay (stdout, true);
	cudaPrintfEnd ();

	hipDeviceSynchronize();
	//-----------------------------------------------------------------------------
	//-----------------------------------------------------------------------------
	//-----------------------------------------------------------------------------
	//-----------------------------------------------------------------------------  copy_back memory from gpu 
	cudaStatus = hipMemcpy(&EFMs[0], &d_EFMs[0], sizeE/*(size_t)sizeof(*d_EFMs)*/, hipMemcpyDeviceToHost);	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//----------------------------------------------------------------------------
Error:
	hipFree(d_network);			
	hipFree(d_EFMs);

	return cudaStatus;
}


//------------------------------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------

int main()
{

	//randPathStab();

	hipError_t cudaStatus;

	char timeStr [9];

	string inFile1, outFile1;
	ifstream *fin;
	ofstream *fout;

	//-------------------- 

	vector<vector<float>> S;
	//vector<NODE> network;
	//NODE_ARRAY network[NumberOfMETABOLITEs];


	_strtime_s(timeStr);
	printf( "\nThe current time is %s \n\n", timeStr);

	// mona
	inFile1 = "inputs\\cho.txt";		//cho	//revTest_1000i				//iAF1260
	outFile1 = "outputs\\cho.txt";				//revTest_1000i

	fin = new ifstream(inFile1.c_str());
	fout = new ofstream(outFile1.c_str());

	//-------------------------------------------------------- create network
	S = readFileS(*fin);

	//network = CreateNetwork(S/*, network*/);
	/*network =*/ CreateNetwork_array(S/*, network*/);

	//--------------------------------------------------------
	cudaStatus = EFM();

	//--------------------------------------------------------
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "EFMFinder failed!");
		return 1;
	}

	//-------------------------------------------------------- results
	//printf();
	//eliminateSameEFMs();
	printEFMs(*fout); 

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	_strtime_s(timeStr);
	printf( "\nThe current time is %s \n\n", timeStr);

	//delete[] network; 

	printf("Press any key to continue...");
	_getch();

	return 0;
}

//----------------------------------------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------------------
// desiging pseudo-recursive functions killing meeeee .. true story :| 
// blew my mind; 
// ---------------------------------------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------------------

